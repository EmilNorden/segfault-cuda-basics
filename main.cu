
#include <hip/hip_runtime.h>
#include <iostream>
#include <array>

__global__ void add_kernel(float *a, float *b, float *result, size_t N) {
    auto index = threadIdx.x;
    if(index < N) {
        result[index] = a[index] + b[index];
    }
}

template <size_t N>
std::array<float, N> add_on_gpu(const std::array<float, N> &a, const std::array<float, N> &b) {
    std::array<float, N> results{};
    // Get some data onto the GPU
    float *device_array_a;
    hipMalloc(&device_array_a, sizeof(float) * N);
    hipMemcpy(device_array_a, a.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    float *device_array_b;
    hipMalloc(&device_array_b, sizeof(float) * N);
    hipMemcpy(device_array_b, b.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    float *device_results;
    hipMalloc(&device_results, sizeof(float) * N);

    // Run our fancy CUDA logic
    add_kernel<<<1, N>>>(device_array_a, device_array_b, device_results, N);

    // Get our data back from the GPU
    hipMemcpy(results.data(), device_results, sizeof(float) * N, hipMemcpyDeviceToHost);

    return results;
}

int main() {

    auto result = add_on_gpu<3>(
            {1,2,3},
            {10,20,30});

    for(auto x : result) {
        std::cout << x << "\n";
    }

    return 0;
}
