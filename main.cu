
#include <hip/hip_runtime.h>
#include <iostream>
#include <array>

template <size_t N>
std::array<float, N> add_on_gpu(const std::array<float, N> &a, const std::array<float, N> &b) {
    std::array<float, N> results{};

    // Get some data onto the GPU

    // Run our fancy CUDA logic

    // Get our data back from the GPU

    return results;
}

int main() {

    auto result = add_on_gpu<3>(
            {1,2,3},
            {10,20,30});

    for(auto x : result) {
        std::cout << x << "\n";
    }

    return 0;
}
